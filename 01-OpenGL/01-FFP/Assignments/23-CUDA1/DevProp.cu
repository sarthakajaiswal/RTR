#include <stdio.h> 

int main(void) 
{
    // function declarations 
    void PrintCUDADeviceProperties(void); 

    // code 
    PrintCUDADeviceProperties(); 
}

void PrintCUDADeviceProperties(void) 
{
    // code 
    printf("CUDA INFORMATION : \n"); 
    printf("======================================================\n"); 

    hipError_t ret_cuda_rt; 
    int dev_count; 
    ret_cuda_rt = hipGetDeviceCount(&devCount); 
    if(ret_cuda_rt != hipSuccess) 
    {
        printf("CUDA Runtime API Error - hipGetDeviceCount() Failed due to %s.\n",
                    hipGetErrorString(ret_cuda_rt)); 
    }
    else if(dev_count == 0) 
    {
        printf("There is no CUDA supported device on this system.\n"); 
        return; 
    }
    else 
    {
        printf("Total Number of CUDA supporting GPU Device/Devices on this system : %d\n", dev_count); 
        for(int i = 0; i < dev_count; ++i) 
        {
            CudaDeviceProp dev_prop; 
            int driverVersion = 0, runtimeVersion = 0; 

            ret_cuda_rt = cudaGetDevicePropeties(&dev_prop, i); 
            if(ret_cuda_rt != hipSuccess) 
            {
                printf("%s in %s at line %d\n", hipGetErrorString(ret_cuda_rt), __FILE__, __LINE__); 
                return; 
            }
            printf("\n"); 

            hipDriverGetVersion(&driverVersion);
            cudaRunTimeGetVersion(&runtimeVersion); 
            
            printf("************** CUDA DRIVER AND RUNTIME INFORMATION *************\n"); 
            printf("=================================================================\n");
            printf("CUDA Driver version                               : %d.%d\n", 
                        driverVersion / 1000, 
                        (driverVersion % 100) / 10
                );
            printf("CUDA runtime version                              : %d.%d\n", 
                        runtimeVersion / 1000, 
                        (runtimeVersion % 100) / 10
                    );
            printf("\n"); 
            printf("=================================================================\n");    
            printf("********* GPU DEVICE GENERAL INFORMATION ***********************\n"); 
            printf("=================================================================\n"); 
            printf("GPU Device Number                                   : %d\n", i); 
            printf("GPU Device Name                                     : %s\n", dev_prop.name); 
            printf("GPU Device Compute Capability                       : %d.%d\n", dev_prop.major, dev_prop.minor); 
            printf("GPU Device Clock Rate                               : %d\n", dev_prop.clockRate); 
            printf("GPU Device Type                                     : "); 
            if(dev_prop.integrated) 
                printf("Integrated ( On-Board )\n"); 
            else 
                printf("Discrete ( Card )\n"); 

            printf("\n"); 
            printf("************* GPU DEVICE MEMORY INFORMATION *********************\n"); 
            printf("====================================================================\n"); 
            printf("GPU Device Total Memory                             : %.0f GB = %.0f MB = %llu bytes\n", 
                        ((float)dev_prop.totalGlobalMem / 1048576.0f) / 1024.0f, 
                        (float)dev_prop.totalGlobalMem / 1048576.0f, 
                        (unsigned long long)dev_prop.totalGlobalMem 
                    ); 
            printf("GPU Device Constant memory                          : %lu bytes\n", 
                        (unsigned long)dev_prop.totalConstMem); 
            printf("GPU Device Shared Memory per SMProcessor            : %lu\n", (unsigned long)dev_prop.sharedMemPerBlock); 
            printf("\n"); 
            printf("************ GPU DEVICE MULTIPROCESSOR INFORMATION *************\n"); 
            printf("===================================================================\n"); 
            printf("GPU Device number of SMProcessors :                 : %lu\n", dev_prop.multiProcessorCount); 
            printf("GPU Device Number of registers per SMProcessor      : %d\n", dev_prop.regsPerBlock); 
            printf("\n"); 
            printf("************ GPU DEVICE THREAD INFORMATION **********************\n"); 
            printf("====================================================================\n"); 
            printf("GPU Device maximum number of threads per SMProcessor: %d\n", dev_prop.maxThreadPerMultiProcessor); 
            printf("GPU Device maximum number of threads per Block      : %d\n", dev_prop.maxThreadPerBlock); 
            printf("GPU Device threads in warp                          : %d\n", dev_prop.warpSize); 
            printf("GPU Device maximum Thread Dimentions                : (%d, %d, %d)\n", 
                        dev_prop.maxThreadDim[0], 
                        dev_prop.maxThreadDim[1], 
                        dev_prop.maxThreadDim[2]
                    );
            printf("GPU Device maximum Grid dimention                   : (%d, %d, %d)\n", 
                        dev_prop.maxGridSize[0], 
                        dev_prop.maxGridSize[1], 
                        dev_prop.maxGridSize[2]
                    );  
            printf("\n"); 
            printf("************** GPU DEVICE DRIVER INFORMATION ********************\n"); 
            printf("====================================================================\n"); 
            printf("GPU Device has ECC support                          : %s\n", 
                        dev_prop.ECCEnabled ? "Enabled" : "Disabled"); 
            
            #if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64) 
                    printf("GPU Device CUDA driver mode (ICC or WDDM )  : %s\n", 
                            dev_prop.tccDriver ? "TCC ( Tesla Compute Cluster Driver )" : "WDDM ( Windows Display Driver Model )"); 
            #endif 
                    printf("***********************************************************************************************************************\n"); 
        }
    }
}
