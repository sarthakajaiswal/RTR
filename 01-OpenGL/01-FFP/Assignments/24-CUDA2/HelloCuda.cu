#include "hip/hip_runtime.h"
// header files 
// standard headers 
#include <stdio.h> 

// cuda headers 
#include <hip/hip_runtime.h> 

// global variables 
const int iNumberOfArrayElements = 5; 

float* hostInput1 = NULL; 
float* hostInput2 = NULL; 
float* hostOutput = NULL; 

float* deviceInput1 = NULL; 
float* deviceInput2 = NULL; 
float* deviceOutput = NULL; 

// CUDA Kernel 
__global__ void vecAddGPU(float* in1, float * in2, float* out, int len) 
{
    // code 
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if(i < len) 
    {
        out[i] = in1[i] + in2[i]; 
    }
}

// entry-point function 
int main(void) 
{
    // function declarations 
    void cleanup(void); 

    // variable declarations 
    int size = iNumberOfArrayElements * sizeof(float); 
    hipError_t result = hipSuccess; 

    // code 
    // host memory allocation 
    hostInput1 = (float*)malloc(size); 
    if(hostInput1 == NULL) 
    {
        printf("Host memory allocation failed for hostInput1 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    hostInput2 = (float*)malloc(size); 
    if(hostInput2 == NULL) 
    {
        printf("Host memory allocation failed for hostInput2 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    } 

    hostOutput = (float*)malloc(size); 
    if(hostOutput == NULL) 
    {
        printf("Host memory allocation failed for hostOutput array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    // filling values into host arrays 
    hostInput1[0] = 101.0f; 
    hostInput1[1] = 202.0f; 
    hostInput1[2] = 303.0f; 
    hostInput1[3] = 404.0f; 
    hostInput1[4] = 505.0f; 

    hostInput2[0] = 201.0f; 
    hostInput2[1] = 202.0f; 
    hostInput2[2] = 203.0f; 
    hostInput2[3] = 204.0f; 
    hostInput2[4] = 205.0f; 

    // device memory allocation 
    result = hipMalloc((void**)&deviceInput1, size); 
    if(result != hipSuccess) 
    {
        printf("Device memory allocation failed for deviceInput1 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    result = hipMalloc((void**)&deviceInput2, size); 
    if(result != hipSuccess) 
    {
        printf("Device memory allocation failed for deviceInput2 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    result = hipMalloc((void**)&deviceOutput, size); 
    if(result != hipSuccess) 
    {
        printf("Device memory allocation failed for deviceOutput array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    // copy data from host arrays into device arrays 
    result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice); 
    if(result != hipSuccess)
    {
        printf("Host to device data copy is failed for deviceInput1 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice); 
    if(result != hipSuccess) 
    {
        printf("Host to device data copy is failed for deviceInput2 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    dim3 dimGrid = dim3(iNumberOfArrayElements, 1, 1); 
    dim3 dimBlock = dim3(1, 1, 1); 

    // cuda kernel for vector addition 
    vecAddGPU <<<dimGrid, dimBlock???(deviceInput1, deviceInput2, deviceOutput, iNumberOfArrayElements); 

    // copy data from device arrays into host arrays 
    result = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost); 
    if(result != hipSuccess) 
    {
        printf("Device to Host data copy is failed for hostOutput.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    // vector addition on host 
    for(int i = 0; i < iNumberOfArrayElements; ++i) 
    {
        printf("%f + %f = %f\n", hostInput1[i], hostInput2[i], hostOutput2[i]); 
    }

    // cleanup 
    cleanup(); 

    return (0); 
}

void cleanup(void) 
{
    // code 
    if(deviceOutput) 
    {
        hipFree(deviceOutput); 
        deviceOutput = NULL; 
    }

    if(deviceInput2)
    {
        hipFree(deviceInput2); 
        deviceInput2 = NULL; 
    }

    if(deviceInput1) 
    {
        hipFree(deviceInput1); 
        deviceInput1 = NULL; 
    }

    if(hostOutput) 
    {
        free(hostOutput); 
        hostOutput = NULL; 
    }

    if(hostInput2) 
    {
        free(hostInput2); 
        hostInput2 = NULL; 
    }

    if(hostInput1) 
    {
        free(hostInput1); 
        hostInput1 = NULL; 
    }
}
