#include "hip/hip_runtime.h"
// header files 
// standard headers 
#include <stdio.h> 

// cuda headers 
#include <hip/hip_runtime.h> 
#include "helper_timer.h" 

// global variable  
// const int iNumberOfArrayElements = 5; 
const int iNumberOfArrayElements = 11444777; 

float* hostInput1 = NULL; 
float* hostInput2 = NULL; 
float* hostOutput = NULL; 
float* gold = NULL; 

float* deviceInput1 = NULL; 
float* deviceInput2 = NULL; 
float* deviceOutput = NULL; 

float timeOnCPU = 0.0f; 
float timeOnGPU = 0.0f; 

// CUDA Kernel 
__global__ void vecAddGPU(float* in1, float* in2, float* output, int len) 
{
    // code 
    int i = blockIdx.x + blockDim.x + threadIdx.x; 

    if(i < len) 
    {
        output[i] = in1[i] + in2[i]; 
    }
}

// entry-point function 
int main(void) 
{
    // function declarations 
    void fillFloatArrayWithRandomNumbers(float*, int); 
    void vecAddCPU(const float*, const float*, float*, int); 
    void cleanup(void); 

    // variable declarations 
    int size = iNumberOfArrayElements * sizeof(float); 
    hipError_t result = hipSuccess; 

    // code 
    // memory allocation on host 
    hostInput1 = (float*)malloc(size); 
    if(hostInput1 == NULL) 
    {
        printf("Host Memory allocation failed for hostInput1 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    hostInput2 = (float*)malloc(size); 
    if(hostInput2 == NULL) 
    {
        printf("Host memory allocation failed for hostInput2 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    hostOutput = (float*)malloc(size); 
    if(hostOutput == NULL) 
    {
        printf("Host memory allocation failed for hostOutput array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    gold = (float*)malloc(size); 
    if(gold == NULL) 
    {
        printf("Host memory allocation falied for gold array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    // filling values into host arrays 
    fillFloatArrayWithRandomNumbers(hostInput1, iNumberOfArrayElements); 
    fillFloatArrayWithRandomNumbers(hostInput2, iNumberOfArrayElements); 

    // device arrays memory allocation 
    result = hipMalloc((void**)&deviceInput1, size); 
    if(result != hipSuccess) 
    {
        printf("Device memory allocation failed for deviceInput1 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    result = hipMalloc((void**)&deviceInput2, size); 
    if(result != hipSuccess) 
    {
        printf("Device memory allocation failed for deviceInput2 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    result = hipMalloc((void**)&deviceOutput, size); 
    if(result != hipSuccess) 
    {
        printf("Device memory allocation failed for deviceOutput array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    // copy data from host arrays into device arrays 
    result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice); 
    if(result != hipSuccess) 
    {
        printf("Host to device data copy is failed for deviceInput1 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice); 
    if(result != hipSuccess) 
    {
        printf("Host to device data copy failed for deviceInput2 array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    // CUDA kernel configuration 
    dim3 dimGrid = dim3((int)ceil((float)iNumberOfArrayElements / 256.0f), 1, 1); 
    dim3 dimBlock = dim3(256, 1, 1); 

    // CUDA kernel for vector addition 
    StopWatchInterface* timer = NULL; 
    sdkCreateTimer(&timer); 
    sdkStartTimer(&timer); 

    vecAddGPU<<<dimGrid, dimBlock>>> (deviceInput1, deviceInput2, deviceOutput, iNumberOfArrayElements); 

    sdkStopTimer(&timer); 
    timeOnGPU = sdkGetTimerValue(&timer); 
    sdkDeleteTimer(&timer); 
    timer = NULL; 

    // copy data from device to host arrays into host arrays 
    result = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost); 
    if(result != hipSuccess) 
    {
        printf("Device to host data copy is failed for hostOutput array.\n"); 
        cleanup(); 
        exit(EXIT_FAILURE); 
    }

    // vector addition on host 
    vecAddCPU(hostInput1, hostInput2, gold, iNumberOfArrayElements); 

    // comparison 
    const float epsilon = 0.000001f; 
    int breakValue = -1; 
    bool bAccuracy = true; 
    for(int i = 0; i < iNumberOfArrayElements; ++i) 
    {
        float val1 = gold[i]; 
        float val2 = hostOutput[i]; 
        if(fabs(val1 - val2) > epsilon) 
        {
            bAccuracy = false; 
            breakValue = i; 
            break; 
        }
    }

    char str[128]; 
    if(bAccuracy == false) 
        sprintf(str, "Comparison of CPU and GPU vector addition is not within accuracy of 0.000001 at array index %d", breakValue); 
    else 
        sprintf(str, "Comparison of CPU and GPU vector addition is within accuracy of 0.000001"); 

    // output 
    printf("Array1 begins from 0th index %.6f to %dth index %.6f\n", 
            hostInput1[0], iNumberOfArrayElements - 1, hostInput1[iNumberOfArrayElements - 1]); 
    printf("Array2 begins from 0th index %.6f to %dth index %.6f\n", 
            hostInput2[0], iNumberOfArrayElements - 1, hostInput2[iNumberOfArrayElements - 1]); 
    printf("CUDA Kernel Grid Dimention = %d,%d,%d and Block Dimention = %d,%d,%d\n", 
            dimGrid.x, dimGrid.y, dimGrid.z, 
            dimBlock.x, dimBlock.y, dimBlock.z); 
    printf("Output array begins from 0th index %.6f to %dth index %.6f\n", 
            hostOutput[0], iNumberOfArrayElements - 1, hostOutput[iNumberOfArrayElements - 1]); 
    printf("Time taken for vector addition on CPU = %.6f\n", timeOnCPU); 
    printf("Time taken for vector addition on GPU = %.6f\n", timeOnGPU); 
    printf("%s\n", str); 

    // cleanup 
    cleanup(); 

    return (0); 
}

void fillFloatArrayWithRandomNumbers(float* arr, int len) 
{
    // code 
    const float fscale = 1.0f / (float)RAND_MAX; 
    for(int i = 0; i < len; ++i) 
    {
        arr[i] = fscale * rand(); 
    }
}

void vecAddCPU(const float* arr1, const float* arr2, float* out, int len) 
{
    // code 
    StopWatchInterface* timer = NULL; 
    sdkCreateTimer(&timer); 
    sdkStartTimer(&timer); 

    for(int i = 0; i < len; ++i) 
    {
        out[i] = arr1[i] + arr2[i]; 
    }

    sdkStopTimer(&timer); 
    timeOnCPU = sdkGetTimerValue(&timer); 
    sdkDeleteTimer(&timer); 
    timer = NULL; 
}

void cleanup(void) 
{
    if(hostOutput)
    {
        free(hostOutput); 
        hostOutput = NULL; 
    }

    if(hostInput2)
    {
        free(hostInput2); 
        hostInput2 = NULL; 
    }

    if(hostInput1) 
    {
        free(hostInput1); 
        hostInput1 = NULL; 
    }

    if(gold) 
    {
        free(gold); 
        gold = NULL; 
    }

    if(deviceOutput)
    {
        hipFree(deviceOutput); 
        deviceOutput = NULL; 
    }

    if(deviceInput2)
    {
        hipFree(deviceInput2); 
        deviceInput2 = NULL; 
    }

    if(deviceInput1) 
    {
        hipFree(deviceInput1); 
        deviceInput1 = NULL; 
    }
}
